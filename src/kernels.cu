#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "doctest/doctest.h"
#include <random>
#include "common.h"

// Wendland C2 kernel

__host__ __device__ __forceinline__
C2::C2(float h_bar) : Kernel(h_bar, 21.f / 2.f * M_1_PI) {};

__host__ __device__ __forceinline__ float C2::w(float q)
{
    const float t1{fmax(0.f, 1.f - q)};
    const float t2{4.f * q + 1.f};
    const float t1_2{t1 * t1};
    return t1_2 * t1_2 * t2;
}

__host__ __device__ __forceinline__ float C2::dw(float q)
{
    const float t1{fmax(0.f, 1.f - q)};
    const float t2{-20.f * q};
    return t1 * t1 * t1 * t2;
}

// Cubic Spline kernel

__host__ __device__ __forceinline__
B3::B3(float h_bar) : Kernel(h_bar, 16.f * M_1_PI) {};
__host__ __device__ __forceinline__ float B3::w(float q)
{
    const float t1{fmax(0.f, 1.f - q)};
    const float t2{sat(0.5f - q)};
    return t1 * t1 * t1 - 4.f * t2 * t2 * t2;
}

__host__ __device__ __forceinline__ float B3::dw(float q)
{
    const float t1{fmax(0.f, 1.f - q)};
    const float t2{sat(0.5f - q)};
    return -3.f * t1 * t1 + 12.f * t2 * t2;
}

// Wendland C6 kernel

__host__ __device__ __forceinline__
W6::W6(float h_bar) : Kernel(h_bar, 1365.f / 64.f * M_1_PI) {};

__host__ __device__ __forceinline__ float W6::w(float q)
{
    const float t1{fmax(0.f, 1.f - q)};
    const float t1_2{t1 * t1};
    const float t1_4{t1_2 * t1_2};
    const float t1_8{t1_4 * t1_4};
    const float q_2{q * q};
    const float t2{32.f * q_2 * q + 25.f * q_2 + 8.f * q + 1.f};
    return t1_8 * t2;
}

__host__ __device__ __forceinline__ float W6::dw(float q)
{
    const float t1{fmax(0.f, 1.f - q)};
    const float t1_2{t1 * t1};
    const float t1_4{t1_2 * t1_2};
    const float t1_7{t1_4 * t1_2 * t1};

    const float t2{(16.f * q * q + 7.f * q + 1.f) * (-22.f * q)};
    return t1_7 * t2;
}

// Double cosine kernel

__host__ __device__ __forceinline__
COS::COS(float h_bar) : Kernel(h_bar, M_PI / (4.f * M_PI * M_PI - 30.f)) {};

__host__ __device__ __forceinline__ float COS::w(float q)
{
    constexpr float PI{static_cast<float>(M_PI)};
    const float pi_s{PI * fmin(1.f, q)};
    return 4.f * cosf(pi_s) + cosf(2.f * pi_s) + 3.f;
}

__host__ __device__ __forceinline__ float COS::dw(float q)
{
    constexpr float PI{static_cast<float>(M_PI)};
    const float pi_s{PI * fmin(1.f, q)};
    constexpr float two_pi{2.f * M_PI};
    return -2.f * two_pi * sinf(pi_s) - sinf(2.f * pi_s) * two_pi;
}

// TESTING ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

template <IsKernel K>
__global__ void _test_kernels(float3 *x, float *ws, float3 *dws, uint N, const K W)
{
    auto i{blockIdx.x * blockDim.x + threadIdx.x};
    if (i >= N)
        return;
    ws[i] = W(x[i]);
    dws[i] = W.nabla(x[i]);
}

// ADD NEW KERNELS TO THE TEMPLATE:

TEST_CASE_TEMPLATE("Kernel function properties", K, B3, C2, W6, COS)
{
    // settings
    const float DEV_HOST_EQ_TOL{0.005};
    const float KERNEL_GRAD_TOL{0.01};
    const uint SAMPLE_COUNT{25};

    // other quantities
    const uint N = SAMPLE_COUNT * SAMPLE_COUNT * SAMPLE_COUNT;
    const float h_bar{1. * SAMPLE_COUNT};
    const K W(h_bar);

    // add interesting values / edge cases to the array, fill the rest pseudo-randomly
    const float EPS = std::numeric_limits<float>::epsilon();
    std::vector<float3> x_rand_h(N);
    x_rand_h[0] = v3(0.f);
    x_rand_h[1] = v3(h_bar);
    x_rand_h[2] = v3(h_bar - EPS);
    x_rand_h[3] = v3(h_bar + EPS);
    x_rand_h[4] = v3(EPS);
    x_rand_h[5] = v3(h_bar + 1e-20);
    x_rand_h[6] = v3(h_bar - 1e-20);
    x_rand_h[7] = v3(1e-20);
    x_rand_h[8] = v3(std::numeric_limits<float>::max());
    std::srand(16142069);
    for (int i{9}; i < N; ++i)
    {
        x_rand_h[i] = v3(
            h_bar * 1.5 * static_cast<float>(std::rand()) / static_cast<float>(RAND_MAX),
            h_bar * 1.5 * static_cast<float>(std::rand()) / static_cast<float>(RAND_MAX),
            h_bar * 1.5 * static_cast<float>(std::rand()) / static_cast<float>(RAND_MAX));
    }

    // also construct positions on a regular grid
    std::vector<float3> x_grid_h(N);
    const float spacing{h_bar * 2. / ((float)SAMPLE_COUNT)};
    uint j{0};
    for (uint xi = 0; xi < SAMPLE_COUNT; ++xi)
    {
        for (uint yi = 0; yi < SAMPLE_COUNT; ++yi)
        {
            for (uint zi = 0; zi < SAMPLE_COUNT; ++zi)
            {
                x_grid_h[j] = v3(-h_bar + spacing / 2) + v3(xi * spacing, yi * spacing, zi * spacing);
                ++j;
            }
        }
    }
    // allocate device memory
    float3 *x_rand_d{nullptr};
    float *ws_d{nullptr};
    float3 *dws_d{nullptr};

    float3 *x_grid_d{nullptr};
    float *reg_ws_d{nullptr};
    float3 *reg_dws_d{nullptr};
    CUDA_CHECK(hipMalloc((void **)&x_rand_d, sizeof(float3) * N));
    CUDA_CHECK(hipMalloc((void **)&ws_d, sizeof(float) * N));
    CUDA_CHECK(hipMalloc((void **)&dws_d, sizeof(float3) * N));

    CUDA_CHECK(hipMalloc((void **)&x_grid_d, sizeof(float3) * N));
    CUDA_CHECK(hipMalloc((void **)&reg_ws_d, sizeof(float) * N));
    CUDA_CHECK(hipMalloc((void **)&reg_dws_d, sizeof(float3) * N));

    // copy pseudo-random positions to the device
    CUDA_CHECK(hipMemcpy(x_rand_d, x_rand_h.data(), sizeof(float3) * N, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(x_grid_d, x_grid_h.data(), sizeof(float3) * N, hipMemcpyHostToDevice));

    // launch the test kernel
    _test_kernels<K><<<BLOCKS(N), BLOCK_SIZE>>>(x_rand_d, ws_d, dws_d, N, W);
    _test_kernels<K><<<BLOCKS(N), BLOCK_SIZE>>>(x_grid_d, reg_ws_d, reg_dws_d, N, W);

    // place results on the heap to avoid stack limits for large tests
    std::vector<float> ws_h(N);
    std::vector<float3> dws_h(N);
    std::vector<float> reg_ws_h(N);
    std::vector<float3> reg_dws_h(N);
    CUDA_CHECK(hipMemcpy(ws_h.data(), ws_d, sizeof(float) * N, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(dws_h.data(), dws_d, sizeof(float3) * N, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(reg_ws_h.data(), reg_ws_d, sizeof(float) * N, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(reg_dws_h.data(), reg_dws_d, sizeof(float3) * N, hipMemcpyDeviceToHost));

    // loop over all results and make assertions
    SUBCASE("Pseudo-random samples: positivity, compact support and similar behaviour on device and host")
    {
        for (int i{0}; i < N; ++i)
        {
            // capture values for more helpful error messages in case a test fails
            CAPTURE(i);
            CAPTURE(x_rand_h[i].x);
            CAPTURE(x_rand_h[i].y);
            CAPTURE(x_rand_h[i].z);

            // check positivity of kernel shape function
            CHECK(ws_h[i] >= doctest::Approx(0.f));
            // check compact support of both kernel shape and its derivative
            if (norm(x_rand_h[i]) >= h_bar)
            {
                CHECK(ws_h[i] == doctest::Approx(0.0f));
                CHECK(norm(dws_h[i]) == doctest::Approx(0.0f));
            }
            // check host-device compatibility
            CHECK(ws_h[i] == doctest::Approx(W(x_rand_h[i])).epsilon(DEV_HOST_EQ_TOL));
            CHECK(dws_h[i].x == doctest::Approx(W.nabla(x_rand_h[i]).x).epsilon(DEV_HOST_EQ_TOL));
            CHECK(dws_h[i].y == doctest::Approx(W.nabla(x_rand_h[i]).y).epsilon(DEV_HOST_EQ_TOL));
            CHECK(dws_h[i].z == doctest::Approx(W.nabla(x_rand_h[i]).z).epsilon(DEV_HOST_EQ_TOL));
        }
        // check that w(0) is not zero, i.e. the call did not go wrong
        CHECK(ws_h[0] > doctest::Approx(0.f));
        // check that dw([0,0,0]) is the zero vector
        CHECK(dws_h[0].x == doctest::Approx(0.f));
        CHECK(dws_h[0].y == doctest::Approx(0.f));
        CHECK(dws_h[0].z == doctest::Approx(0.f));
    }

    SUBCASE("Kernel gradient antisymmetry")
    {
        float kernel_sum{0.};
        float3 kernel_grad_sum{v3(0.)};
        for (int i{0}; i < N; ++i)
        {
            kernel_sum += reg_ws_h[i];
            kernel_grad_sum += reg_dws_h[i];

            // assert kernel gradient symmetry
            uint j{(N - 1) - i};
            CAPTURE(i);
            CHECK(-reg_dws_h[i].x == doctest::Approx(reg_dws_h[j].x));
            CHECK(-reg_dws_h[i].y == doctest::Approx(reg_dws_h[j].y));
            CHECK(-reg_dws_h[i].z == doctest::Approx(reg_dws_h[j].z));
        }

        SUBCASE("Kernel sum is one over volume")
        {
            // the kernel sum should be one over the volume
            CHECK(kernel_sum == doctest::Approx(1. / (spacing * spacing * spacing)).epsilon(DEV_HOST_EQ_TOL));
        }
        SUBCASE("Kernel gradient sum is zero vector")
        {
            // the kernel gradient sum should be zero
            CHECK(kernel_grad_sum.x == doctest::Approx(0.).epsilon(KERNEL_GRAD_TOL));
            CHECK(kernel_grad_sum.y == doctest::Approx(0.).epsilon(KERNEL_GRAD_TOL));
            CHECK(kernel_grad_sum.z == doctest::Approx(0.).epsilon(KERNEL_GRAD_TOL));
        }
    }

    // free the device memory, host vectors are handled by their destructor
    CUDA_CHECK(hipFree(x_rand_d));
    CUDA_CHECK(hipFree(ws_d));
    CUDA_CHECK(hipFree(dws_d));
    CUDA_CHECK(hipFree(x_grid_d));
    CUDA_CHECK(hipFree(reg_ws_d));
    CUDA_CHECK(hipFree(reg_dws_d));
}