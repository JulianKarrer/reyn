#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/scan.h>

#include <random>

#include "datastructure/uniformgrid.cuh"
#include "doctest/doctest.h"
#include <nanobench.h>

/// @brief Called first during uniform grid construction: atomically count the
/// number of particles in each grid cell
__global__ void _count_particles_per_cell(const uint N,
    const float3* __restrict__ x, uint* __restrict__ counts, const uint nx,
    const uint nxny, const float3 bound_min, const float cell_size)
{
    auto i { blockIdx.x * blockDim.x + threadIdx.x };
    if (i >= N)
        return;
    // compute linearized cell index of particle position
    auto index_linear = _index_linear(x[i], bound_min, cell_size, nx, nxny);
    // also, atomically increment the particle count for the cell in the linear
    // list of grid cells
    atomicAdd(&counts[index_linear], 1); // don't use return value
}

/// @brief Called last during uniform grid construction: couting-sort particle
/// indices by obtaining the index into particles of the same cell from the
/// prefix sum and the index within the cell from atomically decrementing the
/// particle counts per cell
__global__ void _counting_sort(const uint N, const float3* __restrict__ x,
    uint* __restrict__ sorted, uint* __restrict__ counts,
    const uint* __restrict__ prefix, const uint nx, const uint nxny,
    const float3 bound_min, const float cell_size)
{
    auto i { blockIdx.x * blockDim.x + threadIdx.x };
    if (i >= N)
        return;
    // recompute linearized particle index
    uint index_linear { _index_linear(x[i], bound_min, cell_size, nx, nxny) };
    // the index to the first particle in the same cell as particle i is given
    // by the number of particles with a lower index, i.e. the prefix sum at i
    uint offset_to_cell { prefix[index_linear] };
    // the offset of i within the cell can be computed by atomically choosing
    // one of the 1..m numbers making up the count of m particles in the same
    // cell, then subtracting one to get an index 0..m-1 for m particles in the
    // same cell
    uint offset_in_cell { atomicSub(&counts[index_linear], 1) - 1 };
    // with both offsets, particle i has a unique spot in the sorted array of
    // particles and can write its index to the corresponding spot
    sorted[offset_to_cell + offset_in_cell] = i;
}

UniformGrid::UniformGrid(const float3 bound_min, const float3 bound_max,
    const float cell_size)
    : // save the cell size of the uniform grid
    _cell_size(cell_size)
    ,
    // the lower bound is offset by a safety margin to make sure all queries
    // in the bounds yield valid indices
    _bound_min(bound_min - v3(cell_size))
    ,
    // after setting bounds and cells size and BEFORE (!) initializing buffers
    // compute their size: compute the number of grid cells along each spatial
    // dimension add one cell size along each axis to account for margins of
    // half a cell
    nxyz(ceil_div(bound_max - _bound_min + v3(2. * cell_size), cell_size))
    ,
    // initialize counts to zero
    counts(nxyz.x * nxyz.y * nxyz.z, 0)
    ,
    // initialization of prefix does not matter, it is overwritten with counts
    prefix(nxyz.x * nxyz.y * nxyz.z)
    ,
    // allocate only minimal memory for now, sorted will be resized to fit the
    // number of particles whenever required
    sorted(1)
{
    // assert that the number of grid cells along each axis is non-negative
    if (nxyz.x <= 0 || nxyz.y <= 0 || nxyz.z <= 0)
        throw std::runtime_error("Negative number of grid cells encountered in "
                                 "construction of uniform grid.");
};

DeviceUniformGrid UniformGrid::update_and_get_pod(const DeviceBuffer<float3>& x)
{
    // get the number of particles
    const uint N { (uint)x.size() };

    // resize the buffer of sorted indices to fit the number of particles, if
    // required initialization does not matter since everything is overwritten
    if (sorted.size() != N)
        sorted.resize(N);

    // - compute the cell index of each particle
    // - linearize it to obtain a pointer into the flat `counts` array
    // - and atomically increment the particle count in the `counts`
    _count_particles_per_cell<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(),
        counts.ptr(), nxyz.x, nxyz.x * nxyz.y, _bound_min, _cell_size);
    CUDA_CHECK(hipGetLastError());

    // copy counts -> prefix
    // this means one copy of counts can be atomically decremented to sort,
    // while another provides offsets by storing the number of particles with a
    // lower linear index (i.e. the result of the exclusive prefix sum or
    // prescan)
    thrust::copy(
        counts.get().begin(), counts.get().end(), prefix.get().begin());

    // then, take a prefix sum of the device vector
    thrust::exclusive_scan(
        prefix.get().begin(), prefix.get().end(), prefix.get().begin());

    // finally, perform a counting sort:
    // the prefix sum is an offset to particles in the same cell, atomicSub on
    // counts then distributes unique offsets on top of that for each particle
    // in the same cell
    _counting_sort<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(), sorted.ptr(),
        counts.ptr(), prefix.ptr(), nxyz.x, nxyz.x * nxyz.y, _bound_min,
        _cell_size);
    CUDA_CHECK(hipGetLastError());

    // pack all relevant pointers and information for queries into a POD struct
    // and return it
    return DeviceUniformGrid {
        .bound_min = _bound_min,
        .cell_size = _cell_size,
        .r_c_2 = _cell_size * _cell_size,
        .nx = nxyz.x,
        .nxny = nxyz.x * nxyz.y,
        .prefix = prefix.ptr(),
        .sorted = sorted.ptr(),
    };
}

// DeviceUniformGrid UniformGrid::update_reorder_and_get_pod(Particles& state)
// {
//     // get the number of particles
//     const uint N { (uint)state.x.size() };

//     // resize the buffer of sorted indices to fit the number of particles, if
//     // required initialization does not matter since everything is
//     overwritten if (sorted.size() != N)
//         sorted.resize(N);

//     // - compute the cell index of each particle
//     // - linearize it to obtain a pointer into the flat `counts` array
//     // - and atomically increment the particle count in the `counts`
//     _count_particles_per_cell<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(),
//         counts.ptr(), nxyz.x, nxyz.x * nxyz.y, _bound_min, _cell_size);
//     CUDA_CHECK(hipGetLastError());

//     // copy counts -> prefix
//     // this means one copy of counts can be atomically decremented to sort,
//     // while another provides offsets by storing the number of particles with
//     a
//     // lower linear index (i.e. the result of the exclusive prefix sum or
//     // prescan)
//     thrust::copy(
//         counts.get().begin(), counts.get().end(), prefix.get().begin());

//     // then, take a prefix sum of the device vector
//     thrust::exclusive_scan(
//         prefix.get().begin(), prefix.get().end(), prefix.get().begin());

//     // finally, perform a counting sort:
//     // the prefix sum is an offset to particles in the same cell, atomicSub
//     on
//     // counts then distributes unique offsets on top of that for each
//     particle
//     // in the same cell
//     _counting_sort<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(), sorted.ptr(),
//         counts.ptr(), prefix.ptr(), nxyz.x, nxyz.x * nxyz.y, _bound_min,
//         _cell_size);
//     CUDA_CHECK(hipGetLastError());

//     // pack all relevant pointers and information for queries into a POD
//     struct
//     // and return it
//     return DeviceUniformGrid {
//         .bound_min = _bound_min,
//         .cell_size = _cell_size,
//         .r_c_2 = _cell_size * _cell_size,
//         .nx = nxyz.x,
//         .nxny = nxyz.x * nxyz.y,
//         .prefix = prefix.ptr(),
//         .sorted = sorted.ptr(),
//     };
// }

// TESTING ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

__global__ void _test_kernel_uniform_grid(const uint N,
    const float3* __restrict__ x, uint* __restrict__ count_out,
    float* __restrict__ len2_out, float3* __restrict__ vec_out,
    const DeviceUniformGrid grid, const float r_c_2)
{
    auto i { blockIdx.x * blockDim.x + threadIdx.x };
    if (i >= N)
        return;

    count_out[i] = grid.ff_nbrs(
        x, i, [r_c_2] __device__(auto i, auto j, auto x_ij, auto x_ij_l2) {
            return (x_ij_l2 <= r_c_2) ? (i == j ? 0u : 1u) : 0u;
        });
    len2_out[i] = grid.ff_nbrs(
        x, i, [r_c_2] __device__(auto i, auto j, auto x_ij, auto x_ij_l2) {
            return (x_ij_l2 <= r_c_2) ? dot(x_ij, x_ij) : 0.f;
        });
    vec_out[i] = grid.ff_nbrs(
        x, i, [r_c_2] __device__(auto i, auto j, auto x_ij, auto x_ij_l2) {
            return (x_ij_l2 <= r_c_2) ? x_ij : v3(0.);
        });
}

__global__ void _test_kernel_uniform_grid_brute_force(const uint N,
    const float3* __restrict__ x, uint* __restrict__ count_out,
    float* __restrict__ len2_out, float3* __restrict__ vec_out,
    const DeviceUniformGrid grid, const float r_c_2)
{
    auto i { blockIdx.x * blockDim.x + threadIdx.x };
    if (i >= N)
        return;

    const float3 x_i { x[i] };
    for (uint j { 0 }; j < N; ++j) {
        const float3 x_ij { x_i - x[j] };
        const float x_ij_l2 { dot(x_ij, x_ij) };
        if (x_ij_l2 <= r_c_2) {
            count_out[i] += i == j ? 0u : 1u;
            len2_out[i] += x_ij_l2;
            vec_out[i] += x_ij;
        }
    }
}

TEST_CASE("Test Uniform Grid")
{
    const uint N { 50000 };
    const float box_size { 1.f };
    const float cell_size { 0.1f };
    const float r_c_2 { cell_size * cell_size };

    /// create a seeded pseudorandom vector of float3 uniformly randomly
    /// distributed in [0; box_size]^3 on the host side
    thrust::host_vector<float3> x_host(N);
    std::mt19937 rng(161420);
    std::uniform_real_distribution<float> uniform_dist(0.f, box_size);
    for (uint i { 0 }; i < N; ++i) {
        x_host[i] = v3(uniform_dist(rng), uniform_dist(rng), uniform_dist(rng));
    }

    // copy the random host-side buffer to the device
    DeviceBuffer<float3> x(N);
    thrust::copy(x_host.begin(), x_host.end(), x.get().begin());

    // create the uniform grid
    UniformGrid uni_grid { UniformGrid(v3(0.), v3(box_size), cell_size) };
    // build the device-side usable POD
    const DeviceUniformGrid grid { uni_grid.update_and_get_pod(x) };

    // allocate buffers for the results
    DeviceBuffer<uint> d_res_count(N, 0);
    DeviceBuffer<uint> d_res_count_bf(N, 0);
    DeviceBuffer<float> d_res_len2(N, 0.f);
    DeviceBuffer<float> d_res_len2_bf(N, 0.f);
    DeviceBuffer<float3> d_res_vec(N, v3(0.f));
    DeviceBuffer<float3> d_res_vec_bf(N, v3(0.f));

    // execute both kernels
    _test_kernel_uniform_grid_brute_force<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(),
        d_res_count_bf.ptr(), d_res_len2_bf.ptr(), d_res_vec_bf.ptr(), grid,
        r_c_2);
    CUDA_CHECK(hipGetLastError());

    _test_kernel_uniform_grid<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(),
        d_res_count.ptr(), d_res_len2.ptr(), d_res_vec.ptr(), grid, r_c_2);
    CUDA_CHECK(hipGetLastError());

    // copy back to host
    thrust::host_vector<uint> h_res_count(
        d_res_count.get().begin(), d_res_count.get().end());
    thrust::host_vector<uint> h_res_count_bf(
        d_res_count_bf.get().begin(), d_res_count_bf.get().end());
    thrust::host_vector<float> h_res_len2(
        d_res_len2.get().begin(), d_res_len2.get().end());
    thrust::host_vector<float> h_res_len2_bf(
        d_res_len2_bf.get().begin(), d_res_len2_bf.get().end());
    thrust::host_vector<float3> h_res_vec(
        d_res_vec.get().begin(), d_res_vec.get().end());
    thrust::host_vector<float3> h_res_vec_bf(
        d_res_vec_bf.get().begin(), d_res_vec_bf.get().end());

    SUBCASE("Uniform Grid Correctness")
    {
        // compare all results
        for (uint i { 0 }; i < N; ++i) {
            // CAPTURE(x.get()[i]);
            CAPTURE(i);
            // make sure there were no out of bounds positions due to potential
            // error in test setup
            CHECK(x_host[i].x >= 0.);
            CHECK(x_host[i].x <= box_size);
            CHECK(x_host[i].y >= 0.);
            CHECK(x_host[i].y <= box_size);
            CHECK(x_host[i].z >= 0.);
            CHECK(x_host[i].z <= box_size);
            // check if the brute-force O(N^2) approach and the uniform grid
            // agree
            CHECK(h_res_count[i] == h_res_count_bf[i]);
            CHECK(h_res_len2[i] == doctest::Approx(h_res_len2_bf[i]));
            CHECK(h_res_vec[i].x == doctest::Approx(h_res_vec_bf[i].x));
            CHECK(h_res_vec[i].y == doctest::Approx(h_res_vec_bf[i].y));
            CHECK(h_res_vec[i].z == doctest::Approx(h_res_vec_bf[i].z));
        }
    }

    // run benchmarks
    ankerl::nanobench::Bench().run("Uniform Grid Construction", [&]() {
        const DeviceUniformGrid grid { uni_grid.update_and_get_pod(x) };
        CUDA_CHECK(hipDeviceSynchronize());
    });

    ankerl::nanobench::Bench().minEpochIterations(5).run(
        "Uniform Grid Query", [&]() {
            _test_kernel_uniform_grid<<<BLOCKS(N), BLOCK_SIZE>>>(N, x.ptr(),
                d_res_count.ptr(), d_res_len2.ptr(), d_res_vec.ptr(), grid,
                r_c_2);
            CUDA_CHECK(hipDeviceSynchronize());
        });
}
