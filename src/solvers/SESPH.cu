#include "hip/hip_runtime.h"
#include "SESPH.cuh"

template <IsKernel K>
__global__ void _compute_densities(float3 *x, float *m, float* rho, uint N, K W){
    // compute index and ensure safety at bounds
    const auto i{blockIdx.x * blockDim.x + threadIdx.x};
    if (i >= N)
        return;
    // read own position only once
    const float3 x_i{x[i]};
    // mass weighted kernel sum gives densities
    float rho_i{0.};
    for (uint j{0}; j<N; ++j){
        rho_i += m[j] * W(x_i - x[j]);
    }
    rho[i] = rho_i;
}

template <IsKernel K>
__global__ void _compute_accelerations_and_integrate(float3 *x, float3 *v,  float *m, float* rho, uint N, const K W, const float k, const float rho_0, const float dt, const float nu, const float h){
    // compute index and ensure safety at bounds
    const auto i{blockIdx.x * blockDim.x + threadIdx.x};
    if (i >= N)
        return;
    // read own buffered values only once, in coalescing fashion
    const float3 x_i{x[i]};
    const float3 v_i{v[i]};
    const float rho_i{rho[i]};
    // compute own pressure once
    const float p_i{fmaxf(0., k * (rho_i/rho_0 - 1.f))};
    // initialize acceleration, resetting previously held value at i
    float3 a_i{v3(0., -9.81, 0.)};
    for (uint j{0}; j<N; ++j){
        const float rho_j{rho[j]};
        const float m_j{m[j]};
        const float3 v_ij{v_i - v[j]};
        const float3 x_j{x[j]};
        const float3 dW{W.nabla(x_i - x_j)};
        const float3 x_ij{x_i - x_j};
        // compute viscous acceleration
        a_i += (
            10.f * nu 
            * m_j/rho_j 
            * dot(v_ij, x_ij) / (dot(x_ij, x_ij) + 0.01*h*h)
            * dW
        );
        // compute pressure at j repeatedly instead of accessing it, since
        // the kernel is memory-bound
        const float p_j{fmaxf(0., k * (rho[j]/rho_0 - 1.f))};
        // compute contribution to pressure acceleration by particle j at particle i
        // note the minus sign!
        a_i -= m_j * (
            p_i/(rho_i * rho_i) + p_j/(rho_j * rho_j) 
        ) * dW;
    }
    // use semi-implicit Euler integration to update velocities and positions
    const float3 v_i_new{v_i + dt * a_i};
    v[i] = v_i_new;
    x[i] += dt * v_i_new;
}

template <IsKernel K>
void SESPH<K>::compute_accelerations(Particles& state, float dt){
    // first, compute densities
    _compute_densities<K><<<BLOCKS(N), BLOCK_SIZE>>>(state.x, state.m, rho, N, W);
    CUDA_CHECK(hipGetLastError());
    // then synchronize
    CUDA_CHECK(hipDeviceSynchronize());
    // and lastly, compute accelerations using these density values
    // note that pressure need not be pre-computed and stored since the kernel is memory-bound, rather compute them on the fly from density at neighbour j
    // also note that viscosity and gravity can be computed on the fly in the inner loop since they require only known velocities and the constant g, so that ∇W_{ij} need only be evaluated once and can be reused for pressure- and non-pressure accelerations
    _compute_accelerations_and_integrate<K><<<BLOCKS(N), BLOCK_SIZE>>>(state.x, state.v, state.m, rho, N, W, k, rho_0, dt, nu, h);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
};

// explicit instantiation in every relevant translation unit
#define X(K) template class SESPH<K>;
FOREACH_KERNEL(X)
#undef X
